#include "include/gpu/gpu_cublas.h"

#ifdef USE_GPU
#ifdef USE_CUBLAS

template<typename T>
void GpuCublas<T>::gemm(bool transA, bool transB, int m, int n, int k, \
        T* a, int lda, T* b, int ldb, T* c, int ldc, T alpha, T beta){

    hipblasHandle_t cublasHandle;
    cublasErrCheck(hipblasCreate(&cublasHandle));

    // warm up
    for(int i = 0; i < Base::_warm_up; ++i){
#ifdef USE_TENSOR_CORE
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                m, n, k, &alpha, 
                                a, HIP_R_32F, lda,
                                b, HIP_R_32F, ldb, &beta,
                                c, HIP_R_32F, ldc,
                                HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
#else
        cublasErrCheck(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                m, n, k, &alpha,
                                a, lda, b, ldb, &beta, c, ldc));
#endif
    }

    Timer<NV> t;
    for(int i = 0; i < Base::_iter_num; ++i){
        t.start();
#ifdef USE_TENSOR_CORE
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, 
                                m, n, k, &alpha, 
                                a, HIP_R_32F, lda,
                                b, HIP_R_32F, ldb, &beta,
                                c, HIP_R_32F, ldc,
                                HIP_R_32F, HIPBLAS_GEMM_DEFAULT));
#else
        cublasErrCheck(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                m, n, k, &alpha,
                                a, lda, b, ldb, &beta, c, ldc));
#endif
        t.end();
    }
    
    Base::_elapsed = t.getAverageTimeMs();
}

template<typename T>
void GpuCublas<T>::operator()(bool transA, bool transB, int m, int n, int k, \
        T* a, int lda, T* b, int ldb, T* c, int ldc, T alpha, T beta){

    T* d_a = nullptr;
    T* d_b = nullptr;
    T* d_c = nullptr;
    
    int size_a = lda * k;
    int size_b = ldb * n;
    int size_c = ldc * n;
    
    cudaErrCheck(hipMalloc((void**)&d_a, size_a * sizeof(T)));
    cudaErrCheck(hipMalloc((void**)&d_b, size_b * sizeof(T)));
    cudaErrCheck(hipMalloc((void**)&d_c, size_c * sizeof(T)));

    cudaErrCheck(hipMemcpy(d_a, a, size_a * sizeof(T), hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(d_b, b, size_b * sizeof(T), hipMemcpyHostToDevice));
    cudaErrCheck(hipMemcpy(d_c, c, size_c * sizeof(T), hipMemcpyHostToDevice));
    
    gemm(transA, transB, m, n, k, d_a, lda, d_b, ldb, d_c, ldc, alpha, beta);
    
    cudaErrCheck(hipMemcpy(c, d_c, size_c * sizeof(T), hipMemcpyDeviceToHost));

    cudaErrCheck(hipFree(d_a));
    cudaErrCheck(hipFree(d_b));
    cudaErrCheck(hipFree(d_c));
}

// template instantiation declarations
template class GpuCublas<float>;

// register CPU_RAW to GEMM Repo;
REGISTER_GEMM(GPU_CUBLAS, GpuCublas);

#endif //USE_CUBLASE
#endif //USE_GPU