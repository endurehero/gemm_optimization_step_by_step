
#include "impl/gpu/gemm_gpu.h"

#ifdef USE_CUBLAS
#include"hipblas.h"

#define CUBLAS_CHECK(condition) \
    do{\
        hipblasStatus_t status = condition;\
        if(condition != HIPBLAS_STATUS_SUCCESS){\
            cout << "line " << __LINE__ << ", cublas error!" << endl;\
            exit(1);\
        }\
    } while(0)

void gemm_gpu(int m, int n, int k, float*a, int lda, float* b, int ldb, float* c, int ldc, float alpha, float beta, float** _C_Dev_Host){

    Timer<NV> t;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;
    
    int size_a = m * k;
    int size_b = k * n;
    int size_c = m * n;
    
    hipMalloc((void**)&d_a, size_a * sizeof(float));
    hipMalloc((void**)&d_b, size_b * sizeof(float));
    hipMalloc((void**)&d_c, size_c * sizeof(float));
    *_C_Dev_Host = (float*)(malloc(size_c * sizeof(float)));
    memset(*_C_Dev_Host, 0, size_c * sizeof(float));
    
    hipMemcpy(d_a, a, size_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size_c * sizeof(float), hipMemcpyHostToDevice);
    
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    t.start();
    CUBLAS_CHECK(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                             m, n, k, &alpha, a, ldb, b, lda, &beta, c, ldc));
    t.end();
    cout << "gpu elapsed time : " << t.elapsed() << " ms,  GFLOPS: " << gflops(2 * m * n * k, t.elapsed()) << endl;
    
    hipMemcpy(*_C_Dev_Host, d_c, size_c * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Gpu processed!" << std::endl;


    CUBLAS_CHECK(hipblasDestroy(handle));
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

#endif