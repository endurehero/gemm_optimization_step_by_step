#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include "common.h"
#include "impl/gpu/gemm_gpu.h"
#include "timer.h"


__global__ void kr_gemm(int m, int n, int k, float* A, int lda, float*B, int ldb, float*C, int ldc, float alpha, float beta){

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;

    int dimx = blockDim.x;
    int dimy = blockDim.y;
    
    for(int r = tidx; r < m; r += dimx){
        for(int c = tidy; c < n; c += dimy){
            float tmp = 0;
            for(int i = 0; i < k; ++i){
                tmp += A[cord(r, i, lda)] * B[cord(i, c, ldb)];
            }

            C[cord(r, c, ldc)] = alpha * tmp + beta * C[cord(r, c, ldc)];
        }
    }
    
}


void gemm_gpu(int m, int n, int k, float*a, int lda, float* b, int ldb, float* c, int ldc, float alpha, float beta, float** _C_Dev_Host){

    Timer<NV> t;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;
    
    int size_a = m * k;
    int size_b = k * n;
    int size_c = m * n;
    
    hipMalloc((void**)&d_a, size_a * sizeof(float));
    hipMalloc((void**)&d_b, size_b * sizeof(float));
    hipMalloc((void**)&d_c, size_c * sizeof(float));
    *_C_Dev_Host = (float*)(malloc(size_c * sizeof(float)));
    memset(*_C_Dev_Host, 0, size_c * sizeof(float));
    
    hipMemcpy(d_a, a, size_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size_c * sizeof(float), hipMemcpyHostToDevice);
    
    
    dim3 Dg(1, 1);
    dim3 Db(16, 16);
    
    t.start();
    kr_gemm<<<Dg, Db>>>(m, n, k, d_a, lda, d_b, ldb, d_c, ldc, alpha, beta);
    t.end();
    cout << "gpu elapsed time : " << t.elapsed() << " ms,  GFLOPS: " << gflops(2 * m * n * k, t.elapsed()) << endl;
    
    hipMemcpy(*_C_Dev_Host, d_c, size_c * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Gpu processed!" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}