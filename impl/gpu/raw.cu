#include "hip/hip_runtime.h"
#include<stdio.h>
#include "impl/gpu/gemm_gpu.h"
#include<hip/hip_runtime.h>

#ifdef USE_GPU_RAW

__global__ void kr_gemm(int m, int n, int k, float* A, int lda, float*B, int ldb, float*C, int ldc, float alpha, float beta){

    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(r < m){
        if(c < n){
            float tmp = 0;
            for(int i = 0; i < k; ++i){
                tmp += A[cord(r, i, lda)] * B[cord(i, c, ldb)];
            }

            C[cord(r, c, ldc)] = alpha * tmp + beta * C[cord(r, c, ldc)];
        }
    }
    
}


void gemm_gpu(int m, int n, int k, float*a, int lda, float* b, int ldb, float* c, int ldc, float alpha, float beta, float** _C_Dev_Host){

    Timer<NV> t;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;
    
    int size_a = m * k;
    int size_b = k * n;
    int size_c = m * n;
    
    hipMalloc((void**)&d_a, size_a * sizeof(float));
    hipMalloc((void**)&d_b, size_b * sizeof(float));
    hipMalloc((void**)&d_c, size_c * sizeof(float));
    *_C_Dev_Host = (float*)(malloc(size_c * sizeof(float)));
    memset(*_C_Dev_Host, 0, size_c * sizeof(float));
    
    hipMemcpy(d_a, a, size_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size_c * sizeof(float), hipMemcpyHostToDevice);
    
    int x_block_size = 32;
    int y_block_size = 16;
    dim3 Dg((m + x_block_size - 1) / x_block_size, (n + y_block_size - 1) / y_block_size);
    dim3 Db(x_block_size, y_block_size);
    
    t.start();
    kr_gemm<<<Dg, Db>>>(m, n, k, d_a, lda, d_b, ldb, d_c, ldc, alpha, beta);
    t.end();
    std::cout << "gpu elapsed time : " << t.elapsed() << " ms,  GFLOPS: " << gflops(2 * m * n * k, t.elapsed()) << std::endl;
    
    hipMemcpy(*_C_Dev_Host, d_c, size_c * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Gpu processed!" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

#endif