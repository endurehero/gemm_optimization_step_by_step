#include "hip/hip_runtime.h"
#include "impl/gpu/gemm_gpu.h"
#include<hip/hip_runtime.h>
#include "stdio.h"

/**
 *   brief: add workload per thread.
 *        if one thread only compute one element in c matrix, it will take 3 instrinctions as following:
 *            #1  load one element in A from shared mem to register file.
 *            #2  load one element in B from shared mem to register file.
 *            #3  FMA.
 *        try to add workload for one thread to increase the ratio of compute / memory load.   
 */

#ifdef USE_GPU_OPT2

#define MACRO_SIZE 64
#define BLOCK_SIZE 16
#define MICRO_SIZE 4

__global__ void kr_gemm(int m, int n, int k, float* a, int lda, float*b, int ldb, float*c, int ldc, float alpha, float beta){

    int block_row = blockIdx.x;
    int block_col = blockIdx.y;

    int row_in_block = threadIdx.x;
    int col_in_block = threadIdx.y;
    
    float c_ans[MICRO_SIZE * MICRO_SIZE];
#pragma unroll
    for(int i = 0; i < MICRO_SIZE * MICRO_SIZE; ++i){
        c_ans[i] = 0.0;
    }
    __shared__ float As[MACRO_SIZE][MACRO_SIZE];
    __shared__ float Bs[MACRO_SIZE][MACRO_SIZE];

    float* sub_c = &c[cord(block_row * MACRO_SIZE, block_col * MACRO_SIZE, ldc)];
    float* sub_a = &a[cord(block_row * MACRO_SIZE, 0, lda)];
    float* sub_b = &b[cord(0, block_col * MACRO_SIZE, ldb)];

    for(int p = 0; p < k / MACRO_SIZE; ++p){
        float* sub_sub_a = &sub_a[cord(0, p * MACRO_SIZE, lda)];
        float* sub_sub_b = &sub_b[cord(p * MACRO_SIZE, 0, ldb)];
        
        
        for(int t = 0; t < MICRO_SIZE; ++t){
            // A first row
            As[col_in_block * MICRO_SIZE + t][row_in_block * MICRO_SIZE] =     sub_sub_a[cord(row_in_block * MICRO_SIZE, col_in_block * MICRO_SIZE + t, lda)];
            // A second row
            As[col_in_block * MICRO_SIZE + t][row_in_block * MICRO_SIZE + 1] = sub_sub_a[cord(row_in_block * MICRO_SIZE + 1, col_in_block * MICRO_SIZE + t, lda)];
            // A third row
            As[col_in_block * MICRO_SIZE + t][row_in_block * MICRO_SIZE + 2] = sub_sub_a[cord(row_in_block * MICRO_SIZE + 2, col_in_block * MICRO_SIZE + t, lda)];
            // A fouth row
            As[col_in_block * MICRO_SIZE + t][row_in_block * MICRO_SIZE + 3] = sub_sub_a[cord(row_in_block * MICRO_SIZE + 3, col_in_block * MICRO_SIZE + t, lda)];
            

            // B first col
            Bs[col_in_block * MICRO_SIZE][row_in_block * MICRO_SIZE + t] =     sub_sub_b[cord(row_in_block * MICRO_SIZE + t, col_in_block * MICRO_SIZE, ldb)];
            // B second col
            Bs[col_in_block * MICRO_SIZE + 1][row_in_block * MICRO_SIZE + t] = sub_sub_b[cord(row_in_block * MICRO_SIZE + t, col_in_block * MICRO_SIZE + 1, ldb)];
            // B third col
            Bs[col_in_block * MICRO_SIZE + 2][row_in_block * MICRO_SIZE + t] = sub_sub_b[cord(row_in_block * MICRO_SIZE + t, col_in_block * MICRO_SIZE + 2, ldb)];
            // B fouth col
            Bs[col_in_block * MICRO_SIZE + 3][row_in_block * MICRO_SIZE + t] = sub_sub_b[cord(row_in_block * MICRO_SIZE + t, col_in_block * MICRO_SIZE + 3, ldb)];

        }

        __syncthreads();
        
        
        int start_row = row_in_block * MICRO_SIZE;
        int start_col = col_in_block * MICRO_SIZE;
        
        for(int i = 0; i < MACRO_SIZE; ++i){
            for(int t = 0; t < MICRO_SIZE; ++t){
                //first row
                c_ans[cord(0, t, MICRO_SIZE)] += As[i][start_row] * Bs[start_col + t][i];
                //second row
                c_ans[cord(1, t, MICRO_SIZE)] += As[i][start_row + 1] * Bs[start_col + t][i];
                //third row
                c_ans[cord(2, t, MICRO_SIZE)] += As[i][start_row + 2] * Bs[start_col + t][i];
                //fouth row
                c_ans[cord(3, t, MICRO_SIZE)] += As[i][start_row + 3] * Bs[start_col + t][i];
                
            }
        }

        __syncthreads();
        
    }

    // store c ans from register file to global mem
    float* sub_sub_c = &sub_c[cord(row_in_block * MICRO_SIZE, col_in_block * MICRO_SIZE, ldc)];
    for(int i = 0; i < MICRO_SIZE; ++i){
        sub_sub_c[cord(0, i, ldc)] = alpha * c_ans[cord(0, i, MICRO_SIZE)] + beta * sub_sub_c[cord(0, i, ldc)];
        sub_sub_c[cord(1, i, ldc)] = alpha * c_ans[cord(1, i, MICRO_SIZE)] + beta * sub_sub_c[cord(1, i, ldc)];
        sub_sub_c[cord(2, i, ldc)] = alpha * c_ans[cord(2, i, MICRO_SIZE)] + beta * sub_sub_c[cord(2, i, ldc)];
        sub_sub_c[cord(3, i, ldc)] = alpha * c_ans[cord(3, i, MICRO_SIZE)] + beta * sub_sub_c[cord(3, i, ldc)];
    }

    
}

void gemm_gpu(int m, int n, int k, float*a, int lda, float* b, int ldb, float* c, int ldc, float alpha, float beta, float** _C_Dev_Host){

    Timer<NV> t;

    float* d_a = nullptr;
    float* d_b = nullptr;
    float* d_c = nullptr;
    
    int size_a = m * k;
    int size_b = k * n;
    int size_c = m * n;
    
    hipMalloc((void**)&d_a, size_a * sizeof(float));
    hipMalloc((void**)&d_b, size_b * sizeof(float));
    hipMalloc((void**)&d_c, size_c * sizeof(float));
    *_C_Dev_Host = (float*)(malloc(size_c * sizeof(float)));
    memset(*_C_Dev_Host, 0, size_c * sizeof(float));
    
    hipMemcpy(d_a, a, size_a * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size_c * sizeof(float), hipMemcpyHostToDevice);

    dim3 Dg((m + MACRO_SIZE - 1) / MACRO_SIZE, (n + MACRO_SIZE - 1) / MACRO_SIZE);
    dim3 Db(BLOCK_SIZE, BLOCK_SIZE);
    
    t.start();
    kr_gemm<<<Dg, Db>>>(m, n, k, d_a, lda, d_b, ldb, d_c, ldc, alpha, beta);
    t.end();
    std::cout << "gpu elapsed time : " << t.elapsed() << " ms,  GFLOPS: " << gflops(2 * m * n * k, t.elapsed()) << std::endl;
    
    hipMemcpy(*_C_Dev_Host, d_c, size_c * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Gpu processed!" << std::endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

#endif